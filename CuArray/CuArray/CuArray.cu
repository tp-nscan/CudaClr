#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <wtypes.h>
#include <stdexcept>
#include "CuArray.h"
#include <sstream>
#include <string>
#include "..\..\Common\ClrUtils.h"

////////////////////////////////////////
// DLL interface
////////////////////////////////////////


BSTR DllTestRuntimeErr()
{
	std::string funcName = "DllTestRuntimeErr";
	char *runtimeErr = "Runtime err msg.";
	std::string err = runtimeErr;
	return RuntimeErrBSTR(err, funcName);
}

BSTR DllTestCudaStatusErr()
{
	std::string funcName = "DllTestCudaStatusErr";
	hipError_t cudaStatus = hipErrorMissingConfiguration;

	return CudaStatusBSTR(cudaStatus, funcName);
}

BSTR DllMallocOnDevice(void **dev_ints, unsigned int bytes)
{
	std::string funcName = "DllMallocOnDevice";
	try
	{
		hipError_t cudaStatus = hipMalloc(dev_ints, bytes);
		if (cudaStatus != hipSuccess) {
			return CudaStatusBSTR(cudaStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


BSTR DllReleaseDevicePtr(void *dev_ptr)
{
	std::string funcName = "DllReleaseDevicePtr";

	try
	{
		hipError_t cudaStatus = hipFree(dev_ptr);
		hipFree(dev_ptr);

		if (cudaStatus != hipSuccess) {
			return CudaStatusBSTR(cudaStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


BSTR DllCopyToDevice(void *dev_ints, const void *host_ints, unsigned int bytes)
{
	std::string funcName = "DllCopyToDevice";
	try
	{
		hipError_t cudaStatus = hipMemcpy(dev_ints, host_ints, bytes, hipMemcpyHostToDevice);

		if (cudaStatus != hipSuccess) {
			return CudaStatusBSTR(cudaStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


BSTR DllCopyFromDevice(void *host_ints, const void *dev_ints, unsigned int bytes)
{
	std::string funcName = "DllCopyFromDevice";
	try
	{
		hipError_t cudaStatus = hipMemcpy(host_ints, dev_ints, bytes, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			return CudaStatusBSTR(cudaStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


BSTR DllCopyDeviceToDevice(void *dest_ints, const void *src_ints, unsigned int bytes)
{
	std::string funcName = "DllCopyDeviceToDevice";
	try
	{
		hipError_t cudaStatus = hipMemcpy(dest_ints, src_ints, bytes, hipMemcpyDeviceToDevice);
		if (cudaStatus != hipSuccess) {
			return CudaStatusBSTR(cudaStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


BSTR DllDeviceSynchronize()
{
	std::string funcName = "DllDeviceSynchronize";
	try
	{
		hipError_t cudaStatus = hipDeviceSynchronize(); 
		if (cudaStatus != hipSuccess) {
			return CudaStatusBSTR(cudaStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


BSTR DllResetDevice()
{
	std::string funcName = "DllResetDevice";
	try
	{
		hipError_t cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			return CudaStatusBSTR(cudaStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}