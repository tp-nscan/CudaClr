#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <wtypes.h>
#include <stdexcept>
#include <sstream>
#include <string>
#include <hipblas.h>
#include "..\..\Common\ClrUtils.h"

extern "C" __declspec(dllexport) BSTR DllMakeCublasHandle(void **dev_handle)
{
	std::string funcName = "DllMakeCublasHandle";
	try
	{
		hipblasHandle_t **gp = (hipblasHandle_t **)dev_handle;
		hipblasStatus_t hipblasStatus_t = hipblasCreate(*gp);

		if (hipblasStatus_t != hipSuccess) {
			return CublasStatusBSTR(hipblasStatus_t, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR DllDestroyCublasHandle(void *dev_handle)
{
	std::string funcName = "DllMakeCublasHandle";
	try
	{
		hipblasHandle_t *gp = (hipblasHandle_t *)dev_handle;
		hipblasStatus_t hipblasStatus_t = hipblasDestroy(*gp);

		if (hipblasStatus_t != hipSuccess) {
			return CublasStatusBSTR(hipblasStatus_t, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}

extern "C" __declspec(dllexport) BSTR DllcublasSgemm(
	void *dev_handle,
	int transa,
	int transb,
	int m,
	int n,
	int k,
	const float *alpha,
	const float *A,
	int lda,
	const float *B,
	int ldb,
	const float *beta,
	float *C,
	int ldc)
{
	std::string funcName = "DllcublasSgemm";
	try
	{
		const float aa = 1.0f;
		const float bb = 0.0f;
		hipblasHandle_t *gp = (hipblasHandle_t *)dev_handle;
		hipblasStatus_t hipblasStatus_t = hipblasSgemm(
			*gp,
			(hipblasOperation_t)transa,
			(hipblasOperation_t)transb,
			m, n, k, &aa, A, lda, B, ldb, &bb, C, ldc);

		if (hipblasStatus_t != hipSuccess) {
			return CublasStatusBSTR(hipblasStatus_t, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}