#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "kernel.h"


hipError_t addWithCuda(int *c, int *a, int *b, unsigned int size);

__global__ void copyKernel(int *out, int *in)
{
	int i = threadIdx.x;
	out[i] = in[i];
}

__global__ void addKernel(int *c, int *a, int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void GolKernel(int *output, int *input, unsigned int span)
{
	for (int i = threadIdx.y + blockDim.y*blockIdx.y; i < span; i += gridDim.y*blockDim.y)
	{
		for (int j = threadIdx.x + blockIdx.x*blockDim.x; j < span; j += blockDim.x*gridDim.x)
		{
			int offset = i * span + j;

			int im = (i - 1 + span) % span;
			int ip = (i + 1) % span;
			int jm = (j - 1 + span) % span;
			int jp = (j + 1) % span;

			int topl = input[im * span + jm];
			int top = input[im * span + j];
			int topr = input[im * span + jp];
			int l = input[i * span + jm];
			int c = input[offset];
			int r = input[i * span + jp];
			int botl = input[ip * span + jm];
			int bot = input[ip * span + j];
			int botr = input[ip * span + jp];

			int sum = topl + top + topr + l + r + botl + bot + botr;

			if (c == 0)
			{
				output[offset] = (sum == 3) ? 1 : 0;
			}
			else
			{
				output[offset] = ((sum == 2) || (sum == 3)) ? 1 : 0;
			}
		}
	}
}


__global__ void Ca9iKernel(int *output, int *input, unsigned int span, float *rands)
{
for (int i = threadIdx.y + blockDim.y*blockIdx.y; i < span; i += gridDim.y*blockDim.y)
{
	for (int j = threadIdx.x + blockIdx.x*blockDim.x; j < span; j += blockDim.x*gridDim.x)
	{
		int offset = i * span + j;

		int im = (i - 1 + span) % span;
		int ip = (i + 1) % span;
		int jm = (j - 1 + span) % span;
		int jp = (j + 1) % span;

		int topl = input[im * span + jm];
		int top = input[im * span + j];
		int topr = input[im * span + jp];
		int l = input[i * span + jm];
		int c = input[offset];
		int r = input[i * span + jp];
		int botl = input[ip * span + jm];
		int bot = input[ip * span + j];
		int botr = input[ip * span + jp];

		int sum = topl + top + topr + l + r + botl + bot + botr;

		if (c == 0)
		{
			output[offset] = (sum == 3) ? 1 : 0;
		}
		else
		{
			output[offset] = ((sum == 2) || (sum == 3)) ? 1 : 0;
		}
	}
}
}


__global__ void Ca9fKernel(float *output, float *input, float *rands, unsigned int span, float step_size, float noise)
{
	for (int i = threadIdx.y + blockDim.y*blockIdx.y; i < span; i += gridDim.y*blockDim.y)
	{
		for (int j = threadIdx.x + blockIdx.x*blockDim.x; j < span; j += blockDim.x*gridDim.x)
		{
			int offset = i * span + j;

			int im = (i - 1 + span) % span;
			int ip = (i + 1) % span;
			int jm = (j - 1 + span) % span;
			int jp = (j + 1) % span;

			float topl = input[im * span + jm];
			float top = input[im * span + j];
			float topr = input[im * span + jp];
			float l = input[i * span + jm];
			float c = input[offset];
			float r = input[i * span + jp];
			float botl = input[ip * span + jm];
			float bot = input[ip * span + j];
			float botr = input[ip * span + jp];

			float sum = c + (topl + top + topr + l + r + botl + bot + botr + rands[offset] * noise) * step_size;

			if (sum > 1.0f)
			{
				sum = 1.0f;
			}
			if (sum < -1.0f)
			{
				sum = -1.0f;
			}
			output[offset] = sum;
		}
	}
}


__global__ void AltKernel(int *data, unsigned int span, int alt, int value)
{
	for (int i = threadIdx.y + blockDim.y*blockIdx.y; i < span; i += gridDim.y*blockDim.y)
	{
		int tw = (i + alt) % 2;

		for (int j = threadIdx.x * 2 + tw + blockIdx.x*blockDim.x; j < span; j += blockDim.x*gridDim.x)
		{
			int offset = i * span + j;
			data[offset] = value;
		}
	}
}

__global__ void AltKernelCopy(int *dataOut, int *dataIn, unsigned int span, int alt, int value)
{
	for (int i = threadIdx.y + blockDim.y*blockIdx.y; i < span; i += gridDim.y*blockDim.y)
	{
		int tw = (i + alt) % 2;

		for (int j = threadIdx.x  + blockIdx.x*blockDim.x; j < span; j += blockDim.x*gridDim.x)
		{
			int offset = i * span + j;

			if ((j + tw) % 2)
			{
				dataOut[offset] = dataIn[offset];
            }
			else
			{
				dataOut[offset] = 1; //value;
			}
		}
	}
}


__global__ void AltIsingKernel(int *data, float *rands, float temp, unsigned int span, int alt)
{
	for (int i = threadIdx.y + blockDim.y*blockIdx.y; i < span; i += gridDim.y*blockDim.y)
	{
		int tw = (i + alt) % 2;

		for (int j = threadIdx.x * 2 + tw + blockIdx.x*blockDim.x; j < span; j += blockDim.x*gridDim.x)
		{
			int offset = i * span + j;

			int im = (i - 1 + span) % span;
			int ip = (i + 1) % span;
			int jm = (j - 1 + span) % span;
			int jp = (j + 1) % span;

			int top = data[im * span + j];
			int l = data[i * span + jm];
			int r = data[i * span + jp];
			int bot = data[ip * span + j];

			int q = (top + l + r + bot);
			float tot = q + rands[i] * temp;
			data[offset] = (tot > 0) ? 1 : -1;
		}
	}
}


__global__ void AltIsingKernelCopy(int *dataOut, int *dataIn, float *rands, float temp, unsigned int span, int alt)
{
	for (int i = threadIdx.y + blockDim.y*blockIdx.y; i < span; i += gridDim.y*blockDim.y)
	{
		int tw = (i + alt) % 2;

		for (int j = threadIdx.x + blockIdx.x*blockDim.x; j < span; j += blockDim.x*gridDim.x)
		{
			int offset = i * span + j;

			if ((j + tw) % 2)
			{
				dataOut[offset] = dataIn[offset];
			}
			else
			{
				int offset = i * span + j;

				int im = (i - 1 + span) % span;
				int ip = (i + 1) % span;
				int jm = (j - 1 + span) % span;
				int jp = (j + 1) % span;

				int top = dataIn[im * span + j];
				int l = dataIn[i * span + jm];
				int r = dataIn[i * span + jp];
				int bot = dataIn[ip * span + j];

				int q = (top + l + r + bot);
				float tot = q + rands[i] * temp;
				dataOut[offset] = (tot > 0) ? 1 : -1;
			}
		}
	}
}



//****************************************************************************
__global__ void device_function_init_YK(double d_t, int* d_spin,
	int* d_bond, double* d_random_data, unsigned int* d_label)
	/*
	Bond connection
	(Komura algorithm)
	*/
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	int la, i, index_min;
	int spin, bond;
	__shared__ double boltz;

	spin = d_spin[index];
	bond = 0;
	index_min = index;
	if (threadIdx.x == 0) {
		boltz = d_t;
	}
	__syncthreads();

	/*------------ Bond connections with left and top sites ---------------*/

	for (i = 0; i<2; i++) {
		if (i == 0)la = (index - 1 + gSpan) % gSpan + ((int)(index / gSpan))*gSpan;
		if (i == 1)la = (index - gSpan + gArea) % gArea;
		if (spin == d_spin[la]) {
			if (boltz < d_random_data[index + i * gArea]) {
				bond |= 0x01 << i;
				index_min = min(index_min, la);
			}
		}
	}

	/*------------ Transfer to global memories ----------------------------*/

	// Transfer "label" and "bond" to a global memory
	d_bond[index] = bond;
	d_label[index] = index_min;
}



//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}

__global__ void IsingKernel(int *data, float *rands, unsigned int span, int alt, float t1, float t2, float t3, float t4)
{
	for (int i = threadIdx.y + blockDim.y*blockIdx.y; i < span; i += gridDim.y*blockDim.y)
	{
		int tw = (i + alt) % 2;

		for (int j = threadIdx.x * 2 + tw + blockIdx.x*blockDim.x; j < span; j += blockDim.x*gridDim.x)
		{
			int offset = i * span + j;

			int im = (i - 1 + span) % span;
			int ip = (i + 1) % span;
			int jm = (j - 1 + span) % span;
			int jp = (j + 1) % span;

			int top = data[im * span + j];
			int l = data[i * span + jm];
			int r = data[i * span + jp];
			int bot = data[ip * span + j];

			int c = data[offset];

			int q = (top + l + r + bot) * c;
			float rr = rands[offset];
			if (q < 0)
			{
				data[offset] = -c;
				return;
			}
			if ((q = 0) && (rr < 0.5))
			{
				data[offset] = -c;
				return;
			}
			if ((q = 2) && (rr < t2))
			{
				data[offset] = -c;
				return;
			}
			if ((q = 4) && (rr < t4))
			{
				data[offset] = -c;
				return;
			}
		}
	}
}
