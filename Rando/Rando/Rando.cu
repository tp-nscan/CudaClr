#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <wtypes.h>
#include <hiprand.h>
#include <stdexcept>
#include <sstream>
#include <string>
#include "..\..\Common\ClrUtils.h"


////////////////////////////////////////
// DLL interface
////////////////////////////////////////

hiprandGenerator_t _curandGenerator;


extern "C" __declspec(dllexport) BSTR DllMakeGenerator64(int seed)
{
	std::string funcName = "DllMakeGenerator64";
	try
	{
		hiprandStatus_t hiprandStatus = hiprandCreateGenerator(&_curandGenerator, HIPRAND_RNG_PSEUDO_XORWOW);
		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			std::string ctx = funcName + ".hiprandCreateGenerator";
			return CurandStatusBSTR(hiprandStatus, ctx);
		}

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::string ctx = funcName + ".cudaDeviceSynchronize_1";
			return CudaStatusBSTR(cudaStatus, ctx);
		}

		hiprandStatus = hiprandSetPseudoRandomGeneratorSeed(_curandGenerator, (long long)seed);
		if (cudaStatus != hipSuccess) {
			std::string ctx = funcName + ".hiprandSetPseudoRandomGeneratorSeed";
			return CudaStatusBSTR(cudaStatus, ctx);
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::string ctx = funcName + ".cudaDeviceSynchronize_2";
			return CudaStatusBSTR(cudaStatus, ctx);
		}

		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR DllMakeGenerator32(int seed)
{
	std::string funcName = "DllMakeGenerator32";
	try
	{
		hiprandStatus_t hiprandStatus = hiprandCreateGenerator(&_curandGenerator, HIPRAND_RNG_PSEUDO_MRG32K3A);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			std::string ctx = funcName + ".hiprandCreateGenerator";
			return CurandStatusBSTR(hiprandStatus, ctx);
		}
		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::string ctx = funcName + ".cudaDeviceSynchronize_1";
			return CudaStatusBSTR(cudaStatus, ctx);
		}

		hiprandStatus = hiprandSetPseudoRandomGeneratorSeed(_curandGenerator, (long long)seed);
		if (cudaStatus != hipSuccess) {
			std::string ctx = funcName + ".hiprandSetPseudoRandomGeneratorSeed";
			return CudaStatusBSTR(cudaStatus, ctx);
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::string ctx = funcName + ".cudaDeviceSynchronize_2";
			return CudaStatusBSTR(cudaStatus, ctx);
		}

		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR DllDestroyGenerator()
{
	std::string funcName = "DllDestroyGenerator";
	try
	{
		hiprandStatus_t hiprandStatus = hiprandDestroyGenerator(_curandGenerator);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR DllMakeRandomInts(unsigned int *devPtr, unsigned int numRands)
{
	std::string funcName = "DllMakeRandomInts";
	try
	{
		hiprandStatus_t hiprandStatus = hiprandGenerateLongLong(_curandGenerator, (unsigned long long *)devPtr, numRands/4);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR
	DllMakeUniformRands(float *devPtr, unsigned int numRands)
{
	std::string funcName = "DllMakeUniformRands";
	try
	{
		hiprandStatus_t hiprandStatus = hiprandGenerateUniform(_curandGenerator, devPtr, numRands);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR DllMakeNormalRands(float *devPtr, unsigned int numRands, float mean, float stdev)
{
	std::string funcName = "DllMakeNormalRands";
	try
	{
		hiprandStatus_t hiprandStatus = hiprandGenerateNormal(_curandGenerator, devPtr, numRands, mean, stdev);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::string ctx = funcName + ".cudaDeviceSynchronize_1";
			return CudaStatusBSTR(cudaStatus, ctx);
		}

		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR DllMakeLogNormalRands(float *devPtr, unsigned int numRands, float mean, float stdev)
{
	std::string funcName = "DllMakeLogNormalRands";
	try
	{
		hiprandStatus_t hiprandStatus =
			hiprandGenerateLogNormal(_curandGenerator, devPtr, numRands, mean, stdev);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}

extern "C" __declspec(dllexport) BSTR DllMakePoissonRands(unsigned int *devPtr, unsigned int numRands, double lambda)
{
	std::string funcName = "DllMakePoissonRands";
	try
	{
		hiprandStatus_t hiprandStatus = hiprandGeneratePoisson(_curandGenerator, devPtr, numRands, lambda);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}