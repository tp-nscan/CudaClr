#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <wtypes.h>
#include <hiprand.h>
#include <stdexcept>
#include <sstream>
#include <string>
#include "Utils.h"


////////////////////////////////////////
// DLL interface
////////////////////////////////////////

extern "C" __declspec(dllexport) BSTR DllMakeGenerator(void **gen, int seed)
{
	std::string funcName = "DllMakeGenerator";
	try
	{
		hiprandGenerator_t **gp = (hiprandGenerator_t **)gen;
		hiprandStatus_t hiprandStatus =
			hiprandCreateGenerator(*gp, HIPRAND_RNG_PSEUDO_XORWOW);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR DllDestroyGenerator(void *gen)
{
	std::string funcName = "DllDestroyGenerator";
	try
	{
		hiprandGenerator_t *gp = (hiprandGenerator_t *)gen;
		hiprandStatus_t hiprandStatus = hiprandDestroyGenerator(*gp);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}


extern "C" __declspec(dllexport) BSTR
DllMakeUniformRands(float *devPtr, void *gen, unsigned int numRands)
{
	std::string funcName = "DllMakeUniformRands";
	try
	{
		hiprandGenerator_t *gp = (hiprandGenerator_t *)gen;
		hiprandStatus_t hiprandStatus =
			hiprandGenerateUniform(*gp, devPtr, numRands);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}

extern "C" __declspec(dllexport) BSTR
DllMakeNormalRands(float *devPtr, void *gen, unsigned int numRands, float mean, float stdev)
{
	std::string funcName = "DllMakeNormalRands";
	try
	{
		hiprandGenerator_t *gp = (hiprandGenerator_t *)gen;
		hiprandStatus_t hiprandStatus =
			hiprandGenerateNormal(*gp, devPtr, numRands, mean, stdev);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}

extern "C" __declspec(dllexport) BSTR
DllMakeLogNormalRands(float *devPtr, void *gen, unsigned int numRands, float mean, float stdev)
{
	std::string funcName = "DllMakeLogNormalRands";
	try
	{
		hiprandGenerator_t *gp = (hiprandGenerator_t *)gen;
		hiprandStatus_t hiprandStatus =
			hiprandGenerateLogNormal(*gp, devPtr, numRands, mean, stdev);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}

extern "C" __declspec(dllexport) BSTR
DllMakePoissonRands(unsigned int *devPtr, void *gen, unsigned int numRands, double lambda)
{
	std::string funcName = "DllMakePoissonRands";
	try
	{
		hiprandGenerator_t *gp = (hiprandGenerator_t *)gen;
		hiprandStatus_t hiprandStatus =
			hiprandGeneratePoisson(*gp, devPtr, numRands, lambda);

		if (hiprandStatus != HIPRAND_STATUS_SUCCESS) {
			return CurandStatusBSTR(hiprandStatus, funcName);
		}
		return BSTR();
	}
	catch (std::runtime_error &e)
	{
		std::string err = e.what();
		return RuntimeErrBSTR(err, funcName);
	}
}